
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdlib>
#include <chrono>
#include <immintrin.h>

using namespace std;			

// Device code
__global__ void VecAdd(float* A, float* B, float* C, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N)    C[i] = A[i] * B[i]*-1.0f;
}
            

			
int main()
{
    int N = 100000000;
	size_t size = N * sizeof(float);
	chrono::high_resolution_clock::time_point t1,t2;
	chrono::duration<double> t;
	__m256 a,b,c,n=_mm256_set1_ps(-1.0f);
    float* d_A;
    float* d_B;
    float* d_C;
	

    // Allocate input vectors h_A and h_B in host memory
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);


    // Initialize input vectors with random stuff
    for(int i=0;i<N;++i)
	{
		h_A[i] = (float)rand();
		h_B[i] = (float)rand();
	}
	std::cout << "Initialized " << N << " float array" << std::endl;

    // Allocate vectors in device memory
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy vectors from host memory to device memory
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);


	// Straight-up single-thread ALU operation on large float array
	t1 = chrono::high_resolution_clock::now();
	for(int i=0;i<N;i++)
	{
        h_C[i] = h_A[i] * h_B[i]*-1.0f;
	}
	t2 = chrono::high_resolution_clock::now();
	t = t2 - t1;
	std::cout << "CPU only :" << 1000.0f*t.count() << " ms" << std::endl;


	// try stuff using only CPU SIMD; Doing 8 32-bit floats at a time, single-thread
	t1 = chrono::high_resolution_clock::now();
	for(int i=0;i<N;i+=8)
	{
		a = _mm256_loadu_ps(h_A+i);
		b = _mm256_loadu_ps(h_B+i);
		c = _mm256_mul_ps(a,b);
		c = _mm256_mul_ps(c,n);
		_mm256_store_ps(h_C+i,c);
	}
	t2 = chrono::high_resolution_clock::now();
	t = t2 - t1;
	std::cout << "CPU SIMD: " << 1000.0f*t.count() << " ms" << std::endl;


	// now with GPU using CUDA. This one on a 3GB 1060GTX
    int threadsPerBlock = 1024;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

	t1 = chrono::high_resolution_clock::now();

    // Invoke kernel
    VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
	t2 = chrono::high_resolution_clock::now();
	t = t2 - t1;
	std::cout << "CUDA   :" << 1000.0f*t.count() << " ms" << std::endl;


    // Copy result from device memory to host memory
    // h_C contains the result in host memory
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	/*
	CUDA appears only slightly faster than single-threaded SIMD operation. The transfer
	from GPU memory to system memory with cudaMemcpy is very slow, but the floating-point
	operation on the array itself is very fast.
	*/



    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
            
    // Free host memory
    free(h_A);
	free(h_B);
	free(h_C);
}